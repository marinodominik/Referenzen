#include "hip/hip_runtime.h"
#include "lsqrCUDAcuSparseKernel.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include ""
#include <hipsparse.h>

#define BLOCK_SIZE 32            //max threads in a block

__global__ void sqaure_vector(const double *vector, double *tmp, const int size);
__global__ void norm2(const double *in_data, double *result, int size);
__global__ void add_subtract_vector(double *a, double *b, bool operation, int size);  
__global__ void scalar_vector(double *in_data, const double scalar, const int size);
__global__ void matrix_vector_multiplication(const int n_rows, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result);
//__global__ void matrix_vector_multiplication_sh(const int n_row, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result);


inline unsigned int div_up(unsigned int numerator, unsigned int denominator) { //numerator = zähler, denumerator = nenner
	unsigned int result = numerator / denominator;
	if (numerator % denominator) ++result;
    return result;
    
}


GPUMatrix transpose_matrix(GPUMatrix A) {
    GPUMatrix A_transpose = matrix_alloc_sparse_gpu(A.height, A.width, A.elementSize, A.rowSize, A.columnSize);
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    kernelCheck(__LINE__);

    size_t tempInt;
    double *buffer;

    hipsparseCsr2cscEx2_bufferSize(handle, A.height, A.width, A.elementSize,
                                  A.elements, A.csrRow, A.csrCol,
                                  A_transpose.elements, A_transpose.csrCol,A_transpose.csrRow, 
                                  HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &tempInt);
    kernelCheck(__LINE__);
    hipMalloc(&buffer, tempInt);
    kernelCheck(__LINE__);
    hipsparseCsr2cscEx2(handle, A.height, A.width, A.elementSize,
                       A.elements, A.csrRow, A.csrCol,
                       A_transpose.elements, A_transpose.csrRow, A_transpose.csrCol, 
                       HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);
    kernelCheck(__LINE__);
    return A_transpose;
}



/*
<<<<<<<<<<-------------------- NORM ----------------------------->>>>>>>>>>>>>>
*/

__global__ void sqaure_vector(const double *vector, double *tmp, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= size) { 
        tmp[i] = 0;
    } else {
        tmp[i] = vector[i] * vector[i];
    }

    __syncthreads();
}


__global__ void norm2(const double *in_data, double *result,int size) {
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid < size){
        sdata[tid] = in_data[i];        //load global data in sh_memory
    }else{
        sdata[tid] = 0; 
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    //thread 0 writes in result back to global memory
    if (tid == 0) {
        result[blockIdx.x] = sdata[0]; 
    }
}


double getNorm2(const GPUMatrix denseVector) {
    GPUMatrix tmp = matrix_alloc_gpu(denseVector.height, denseVector.width);

    int grids = div_up(denseVector.height, BLOCK_SIZE * BLOCK_SIZE);

    double *result;
    hipMalloc(&result, grids * sizeof(double));
    
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    int sh_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(double);
    
    sqaure_vector<<<grids, dimBlock>>>(denseVector.elements, tmp.elements, tmp.height * tmp.width); 
    norm2<<<grids, dimBlock, sh_memory_size>>>(tmp.elements, result, tmp.height * tmp.width);

    
    double *values = new double[grids]; 
    hipMemcpy(values, result, grids * sizeof(double), hipMemcpyDeviceToHost);

    double norm = 0.0;
    for (int i= 0; i< grids; i++) {
        norm += values[i];
    }

    matrix_free_gpu(tmp);
    delete[] values;
    hipFree(result);

    return sqrt(norm);
}


/*
<<<<<<<<<<-------------------- END NORM ----------------------------->>>>>>>>>>>>>>>>>
*/





/*
<<<<<<<<<<-------------------- ADDITION AND SUBSTRACTION ----------------------------->>>>>>>>>>>>>>>>>
*/

void get_add_subtract_vector(GPUMatrix denseA, GPUMatrix denseB, bool operation) {
    int grids = div_up(denseA.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);

    add_subtract_vector<<<grids, dimBlock>>>(denseA.elements, denseB.elements, operation, denseA.width * denseA.height);
}



__global__ void add_subtract_vector(double *a, double *b, bool operation, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    //check if index out of range of vector
    if(i >= size) return;

    if(operation == true) {
        a[i] = a[i] + b[i];

    } else {
        a[i] = a[i] - b[i];
    }
    __syncthreads();
}


/*
<<<<<<<<<<-------------------- END ADDITON AND SUBSTRACTION ----------------------------->>>>>>>>>>>>>>>>>
*/




/*
<<<<<<<<<<-------------------- MULTIPLY SCALAR ----------------------------->>>>>>>>>>>>>>>>>
*/

void multiply_scalar_vector(GPUMatrix vector, const double scalar) {
    int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);

    scalar_vector<<<grids, dimBlock>>>(vector.elements, scalar, vector.height * vector.width);
}


__global__ void scalar_vector(double *in_data, const double scalar, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        in_data[i] = scalar * in_data[i];
    }
    __syncthreads();
}
/*
<<<<<<<<<<-------------------- END MULTIPLICATION SCALAR ----------------------------->>>>>>>>>>>>>>>>>
*/




/*
<<<<<<<<<<-------------------- CSR MATRIX MULTIPLY WITH DENSE VECTOR ----------------------------->>>>>>>>>>>>>>>>>
*/

__global__ void matrix_vector_multiplication(const int n_rows, const double *elements, 
                                            const int *rowPtr, const int *colIdx, const double *x, double *result) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n_rows) {
        const int row_start = rowPtr[row];
        const int row_end = rowPtr[row + 1];

        double sum = 0.0;
        for (int idx = row_start; idx < row_end; idx++) {
            int col = colIdx[idx];
            sum += elements[idx] * x[col];
        }
        result[row] = sum;
    }
    __syncthreads();
}

// __global__ void matrix_vector_multiplication_sh(const int n_row, const double *elements, const int *rowPtr, const int *colIdx, const double *x, double *result) {
    
// }


// GPUMatrix get_csr_matrix_vector_multiplication_sh(const GPUMatrix matrix, const GPUMatrix vector, GPUMatrix result) {
//     int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
//     dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
//     int sh_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(double);

//     matrix_vector_multiplication_sh<<<grids, dimBlock, sh_memory_size>>>(matrix.height, matrix.elements, matrix.csrRow, matrix.csrCol, vector.elements, result.elements);
// }



void get_csr_matrix_vector_multiplication(const GPUMatrix matrix, const GPUMatrix vector, GPUMatrix result) {
    int grids = div_up(vector.height, BLOCK_SIZE * BLOCK_SIZE);
    kernelCheck(__LINE__);
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    kernelCheck(__LINE__);
    matrix_vector_multiplication<<<grids, dimBlock>>>(matrix.height, matrix.elements, matrix.csrRow, matrix.csrCol, vector.elements, result.elements);
    kernelCheck(__LINE__);
}


/*
<<<<<<<<<<-------------------- END MATRIX VECTOR MULTIPLICATION ----------------------------->>>>>>>>>>>>>>>>>
*/

 
	
GPUMatrix lsqr_algrithm(const GPUMatrix &A, const GPUMatrix &b, const int max_iters, const double ebs) {
    //printf("-----------------------------INIZIALZATION--------------------------\n");
    int i = 0;
    double curr_err;
    double prev_err = DBL_MAX;
    kernelCheck(__LINE__);
    GPUMatrix x = matrix_alloc_gpu(b.height, b.width);
    kernelCheck(__LINE__);
    GPUMatrix u = matrix_alloc_gpu(b.height, b.width);
    kernelCheck(__LINE__);
    GPUMatrix v = matrix_alloc_gpu(b.height, b.width);
    kernelCheck(__LINE__);
    GPUMatrix w = matrix_alloc_gpu(b.height, b.width);
    kernelCheck(__LINE__);
    GPUMatrix A_transpose = transpose_matrix(A);
    kernelCheck(__LINE__);

    hipEvent_t evStart, evStop;
	hipEventCreate(&evStart);
 	hipEventCreate(&evStop);
	hipEventRecord(evStart, 0);
    

    //<<<<<< -------------- INIZIALIZATION PART -------------------->>>>>>>>>>>>>
    //beta = norm(b);
    double beta = getNorm2(b);
    kernelCheck(__LINE__);
    //printf("beta: %lf\n", beta);

    //u = b/beta;
    hipMemcpy(u.elements, b.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
    kernelCheck(__LINE__);
    multiply_scalar_vector(u, 1 / beta);
    kernelCheck(__LINE__);

    //v = A' * u;
    get_csr_matrix_vector_multiplication(A_transpose, u, v);
    kernelCheck(__LINE__);

    //alpha = norm(v);
    double alpha = getNorm2(v);
    //printf("alpha: %lf\n", alpha);

    //v = v/alpha;
    multiply_scalar_vector(v, 1 / alpha);
    kernelCheck(__LINE__);


    //phi_hat = beta;
    double phi_hat = beta;
    //printf("phi_hat: %lf\n", phi_hat);

    //rho_hat = alpha;
    double rho_hat = alpha;
    //printf("rho_hat: %lf\n", rho_hat);

    //w = v
    hipMemcpy(w.elements, v.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
    kernelCheck(__LINE__);
    //printf("\n------------------------- GETTING IN THE LOOP-----------------------\n");

    GPUMatrix tmp = matrix_alloc_gpu(b.height, b.width);
    for (; i < max_iters; i++) {
        //3a)
        //u = A * v - alpha * u;
        get_csr_matrix_vector_multiplication(A, v, tmp);
        kernelCheck(__LINE__);
        

        multiply_scalar_vector(u, alpha);
        kernelCheck(__LINE__);
        get_add_subtract_vector(tmp, u, false);
        kernelCheck(__LINE__);

        //beta = norm(u);
        beta = getNorm2(tmp);
        kernelCheck(__LINE__);
        //printf("beta: %lf\n", beta);
        
        
        //u = u / beta;
        hipMemcpy(u.elements, tmp.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
        kernelCheck(__LINE__);
        multiply_scalar_vector(u, 1/beta);
        kernelCheck(__LINE__);

        //3b)
        // v = A' * u - beta * v;
        get_csr_matrix_vector_multiplication(A_transpose, u, tmp);
        kernelCheck(__LINE__);
        multiply_scalar_vector(v, beta);
        kernelCheck(__LINE__);
        get_add_subtract_vector(tmp, v, false);
        kernelCheck(__LINE__);


        //alpha = norm(v);
        alpha = getNorm2(tmp);
        kernelCheck(__LINE__);
        //printf("alpha: %lf\n", alpha);


        //v = v / alpha;
        hipMemcpy(v.elements, tmp.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
        kernelCheck(__LINE__);
        multiply_scalar_vector(v, 1/alpha);
        kernelCheck(__LINE__);


        //rho = sqrt(rho_hat^2 + beta^2);
        double rho = sqrt(rho_hat * rho_hat + beta * beta);
        //c = rho_hat / rho;
        double c = rho_hat / rho;
        // s = beta / rho;
        double s = beta / rho;
        //theta = s * alpha;
        double theta = s * alpha;
        //rho_hat = -c * alpha;
        rho_hat = -c * alpha;
        //phi = c * phi_hat;
        double phi = c * phi_hat;
        //phi_hat = s * phi_hat;
        phi_hat = s * phi_hat;
        //printf("rho: %lf, c: %lf, s: %lf, theta: %lf, rho_hat: %lf, phi: %lf, phi_hat: %lf\n", rho, c, s, theta, rho_hat, phi, phi_hat);

        //5
        //x = x + (phi / rho) * w;
        hipMemcpy(tmp.elements, w.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
        kernelCheck(__LINE__);
        multiply_scalar_vector(tmp, phi / rho);
        kernelCheck(__LINE__);
        get_add_subtract_vector(x, tmp, true);
        kernelCheck(__LINE__);
        

        // w = v - (theta / rho) * w;
        multiply_scalar_vector(w, (theta / rho));
        kernelCheck(__LINE__);
        hipMemcpy(tmp.elements, v.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
        kernelCheck(__LINE__);
        get_add_subtract_vector(v, w, false);
        kernelCheck(__LINE__);


        hipMemcpy(w.elements, v.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
        kernelCheck(__LINE__);
        hipMemcpy(v.elements, tmp.elements, b.height * sizeof(double), hipMemcpyDeviceToDevice);
        kernelCheck(__LINE__);
        
        get_csr_matrix_vector_multiplication(A, x, tmp);
        kernelCheck(__LINE__);
        get_add_subtract_vector(tmp, b, false);
        kernelCheck(__LINE__);
        curr_err = getNorm2(tmp);
        kernelCheck(__LINE__);
        //printf("curr_err: %lf\n", curr_err);

        double improvment = prev_err - curr_err;
        if(i%200==0)printf("line: %d size of error: %.6f \n",i,curr_err);
        if (curr_err < ebs) break;
        //printf("i: %d ------------------------NEXT ITERATION -----------------------\n", i); 
    }

    printf("LSQR using kernels finished.\n Iterations num: %d\n Size of error: %.6f\n",i,curr_err);
    hipEventRecord(evStop, 0);
    hipEventSynchronize(evStop);
    float elapsedTime_ms;
    hipEventElapsedTime(&elapsedTime_ms, evStart, evStop);
    printf("LSQR using our kernels took: %f ms\n", elapsedTime_ms);
    hipEventDestroy(evStart);
    hipEventDestroy(evStop);

    matrix_free_gpu(tmp);
    matrix_free_gpu(u);
    matrix_free_gpu(v);
    matrix_free_gpu(w);
    
    //printf("----------------------END KERNEL------------------------\n\n\n");
    return x; 
}


CPUMatrix sparseLSQR_with_kernels(const CPUMatrix &A, const CPUMatrix &b, const int max_iters, const double ebs) {
    kernelCheck(__LINE__);
    CPUMatrix resultCPU = matrix_alloc_cpu(b.height, b.width);
    kernelCheck(__LINE__);
    GPUMatrix resultGPU = matrix_alloc_gpu(b.height, b.width);
    kernelCheck(__LINE__);

    GPUMatrix A_gpu = matrix_alloc_sparse_gpu(A.height, A.width, A.elementSize, A.rowSize, A.columnSize);
    kernelCheck(__LINE__);
    GPUMatrix b_gpu = matrix_alloc_gpu(b.height, b.width);
    kernelCheck(__LINE__);

    /* upload Matrix, vector */
    matrix_upload_cuSparse(A, A_gpu);
    kernelCheck(__LINE__);
    matrix_upload(b, b_gpu);
    kernelCheck(__LINE__);
    resultGPU = lsqr_algrithm(A_gpu, b_gpu, max_iters, ebs);
    /* Download result */
    matrix_download(resultGPU, resultCPU);
    kernelCheck(__LINE__);
    
    /* free GPU memory */
    matrix_free_sparse_gpu(A_gpu);
    matrix_free_gpu(b_gpu);
    matrix_free_gpu(resultGPU);

    return resultCPU;
}


void printVectorKernel(int iteration,GPUMatrix x, const char* name){
	printf("%s: ",name);
	CPUMatrix tempCPUMatrix = matrix_alloc_cpu(x.height, x.width);
	matrix_download(x ,tempCPUMatrix);
	//printf("iteration number: %d\n", iteration);
	for(int i = 0; i < tempCPUMatrix.height; i++){
		printf("%lf ", tempCPUMatrix.elements[i]);
	}
	printf("\n");
}

void printValuesKernel(GPUMatrix x, const char *name) {
    printf("%s: ",name);
	CPUMatrix tempCPUMatrix = matrix_alloc_sparse_cpu(x.height, x.width, x.elementSize, x.rowSize, x.columnSize);
    matrix_download_cuSparse(x ,tempCPUMatrix);
    

    for(int i = 0; i < x.elementSize; i++){
		printf("%lf ", tempCPUMatrix.elements[i]);
    }
    printf("\n Row:");

    for(int i = 0; i < x.rowSize; i++){
		printf("%d ", tempCPUMatrix.csrRow[i]);
    }
    printf("\n Col:");
    for(int i = 0; i < x.columnSize; i++){
		printf("%d ", tempCPUMatrix.csrCol[i]);
    }
    printf("\n");
}


void kernelCheck(int line){
	const hipError_t err = hipGetLastError();                            
	if (err != hipSuccess) {                                              
    	const char *const err_str = hipGetErrorString(err);               
    	std::cerr << "Cuda error in " << __FILE__ << ":" << line - 1   
            << ": " << err_str << " (" << err << ")" << std::endl;   
            exit(EXIT_FAILURE);                                                                    
	}
}